#include "hip/hip_runtime.h"
//   Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <string>

#include "paddle/fluid/operators/fused/fused_seqpool_cvm_with_conv_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

#define GET_BLOCK(N) \
  ((N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS)

#define CUDA_KERNEL_LOOP(i, n)                                  \
  for (auto i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

// normal
template <typename T>
__global__ void FusedSeqpoolWithConvKernelNormal(const size_t N,
                                                 T **input_values,
                                                 T *seqpool_output_values,
                                                 const size_t *lods_values,
                                                 const int batch_size,
                                                 const int embedding_size,
                                                 const float pad_value) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];

    double val = pad_value;
    for (auto k = start; k < end; ++k) {
      val += *(input_values[x] + k * embedding_size + offset);
    }
    seqpool_output_values[i] = val;
  }
}

// Filter
template <typename T>
__global__ void FusedSeqpoolWithConvKernelFilter(const size_t N,
                                                 T **input_values,
                                                 T *seqpool_output_values,
                                                 const size_t *lods_values,
                                                 const int batch_size,
                                                 const int embedding_size,
                                                 const float pad_value,
                                                 const float show_coeff,
                                                 const float clk_coeff,
                                                 const float threshold) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];

    double val = pad_value;
    for (auto k = start; k < end; ++k) {
      T *in = (input_values[x] + k * embedding_size);
      T &show = in[0];
      T &click = in[1];
      if ((show - click) * show_coeff + click * clk_coeff < threshold) {
        continue;
      }
      val += in[offset];
    }
    seqpool_output_values[i] = val;
  }
}

// normal & expand slot's feasign
template <typename T>
__global__ void FusedSeqpoolWithConvKernelNormalEmbedxConcate(
    const size_t N,
    T **input_values,
    T *seqpool_output_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const float pad_value,
    const int embedx_concate_size) {
  int concat_embedding_size = embedding_size * embedx_concate_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int concate_index = concat_offset / embedding_size;  // concat id
    int offset = concat_offset % embedding_size;         // embedx id
    int x = key / batch_size;                            // slot id
    int y = key % batch_size;                            // ins id
    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];

    auto concat_end_pos = start + concate_index + 1;
    if (concat_end_pos > end) {
      concat_end_pos = end;
    }
    double val = pad_value;
    for (auto k = start + concate_index; k < concat_end_pos; ++k) {
      val += *(input_values[x] + k * embedding_size + offset);
    }
    seqpool_output_values[i] = val;
  }
}

// join only need show input
template <typename T>
__global__ void FusedCVMWithConvKernelNormal(const size_t N,
                                             T **output_values,
                                             const T *seqpool_output_values,
                                             const int batch_size,
                                             const int embedding_size) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    const T *in = &seqpool_output_values[(x * batch_size + y) * embedding_size];
    T *out = output_values[x] + y * embedding_size + offset;

    if (offset == 0) {  // show
      *out = log(in[0] + 1);
    } else if (offset == 1) {  // click
      *out = log(in[1] + 1);
    } else if (offset == 2) {  // conv
      *out = log(in[2] + 1) - log(in[1] + 1);
    } else {  // filter show, offset - 1
      *out = in[offset];
    }
  }
}

// join only need show input, and expand slot's feasign
template <typename T>
__global__ void FusedCVMWithConvKernelNormalConcate(
    const size_t N,
    T **output_values,
    const T *seqpool_output_values,
    const int batch_size,
    const int embedding_size,
    const int embedx_concate_size) {
  int concat_embedding_size = embedding_size * embedx_concate_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int k = concat_offset / embedding_size;  // concat id
    int offset = concat_offset % embedding_size;

    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    const T *in_ptr =
        &seqpool_output_values[(x * batch_size + y) * concat_embedding_size +
                               k * embedding_size];
    T *out = output_values[x] + y * concat_embedding_size + k * embedding_size +
             offset;
    if (offset == 0) {  // show
      *out = log(in_ptr[0] + 1);
    } else if (offset == 1) {  // click
      *out = log(in_ptr[1] + 1);
    } else if (offset == 2) {  // conv
      *out = log(in_ptr[2] + 1) - log(in_ptr[1] + 1);
    } else {
      *out = in_ptr[offset];
    }
  }
}

// join only need show input
template <typename T>
__global__ void FusedCVMWithConvKernelWithOutShow(
    const size_t N,
    T **output_values,
    const T *seqpool_output_values,
    const int batch_size,
    const int embedding_size,
    const int noclk_embedding_size) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / noclk_embedding_size;
    int offset = i % noclk_embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    const T *in = &seqpool_output_values[(x * batch_size + y) * embedding_size];
    T *out = output_values[x] + y * noclk_embedding_size + offset;
    if (offset == 0) {  // click
      *out = log(in[1] + 1);
    } else if (offset == 1) {  // conv
      *out = log(in[2] + 1) - log(in[1] + 1);
    } else {  // skip show in offset + 1
      *out = in[offset + 1];
    }
  }
}

// join only need show input, and expand slot's feasign
template <typename T>
__global__ void FusedCVMWithConvKernelWithOutShowConcate(
    const size_t N,
    T **output_values,
    const T *seqpool_output_values,
    const int batch_size,
    const int embedding_size,
    const int noclk_embedding_size,
    const int embedx_concate_size) {
  int concat_embedding_size = embedx_concate_size * noclk_embedding_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int k = concat_offset / noclk_embedding_size;  // concat id
    int offset = concat_offset % noclk_embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    const T *in_ptr =
        &seqpool_output_values[(x * batch_size + y) * embedding_size *
                                   embedx_concate_size +
                               k * embedding_size];
    T *out = output_values[x] + y * concat_embedding_size +
             k * noclk_embedding_size + offset;
    if (offset == 0) {  // click
      *out = log(in_ptr[1] + 1);
    } else if (offset == 1) {  // conv
      *out = log(in_ptr[2] + 1) - log(in_ptr[1] + 1);
    } else {  // filter show, offset + 1
      *out = in_ptr[offset + 1];
    }
  }
}

// update not need show click input
template <typename T>
__global__ void FusedCVMWithConvKernelNoCVM(const size_t N,
                                            T **output_values,
                                            const T *seqpool_output_values,
                                            const int batch_size,
                                            const int no_cvm_embedding_size,
                                            const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / no_cvm_embedding_size;
    int offset = i % no_cvm_embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    const T *in =
        &seqpool_output_values[key * (no_cvm_embedding_size + cvm_offset)];
    // no cvm
    *(output_values[x] + y * no_cvm_embedding_size + offset) =
        in[offset + cvm_offset];
  }
}

// update not need show click input, expand slot's feasign
template <typename T>
__global__ void FusedCVMWithConvKernelNoCVMConcate(
    const size_t N,
    T **output_values,
    const T *seqpool_output_values,
    const int batch_size,
    const int no_cvm_embedding_size,
    const int cvm_offset,
    const int embedx_concate_size) {
  int embedding_size = (no_cvm_embedding_size + cvm_offset);
  int concat_embedding_size = embedx_concate_size * no_cvm_embedding_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int k = concat_offset / no_cvm_embedding_size;  // concat id
    int offset = concat_offset % no_cvm_embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    const T *in =
        &seqpool_output_values[key * embedding_size * embedx_concate_size +
                               k * embedding_size];

    // no cvm
    *(output_values[x] + y * concat_embedding_size + k * no_cvm_embedding_size +
      offset) = in[offset + cvm_offset];
  }
}

template <typename T>
void FusedSeqpoolCVMWithConv(const paddle::platform::Place &place,
                             const std::vector<const T *> &input_data,
                             const std::vector<T *> &output_data,
                             T *gpu_seqpool_output_values,
                             const size_t *lods_values,
                             const int batch_size,
                             const int slot_num,
                             const int embedding_size,
                             const float padding_value,
                             const bool use_cvm,
                             float need_filter,
                             float show_coeff,
                             float clk_coeff,
                             float threshold,
                             const int cvm_offset,
                             bool show_filter,
                             const int embedx_concate_size) {
  auto stream = dynamic_cast<phi::GPUContext *>(
                    platform::DeviceContextPool::Instance().Get(place))
                    ->stream();

  size_t total_ptr_len = input_data.size() + output_data.size();
  auto temp_ptr = memory::AllocShared(place, total_ptr_len * sizeof(void *));
  void *ptr = temp_ptr->ptr();

  T **gpu_input_values = reinterpret_cast<T **>(temp_ptr->ptr());
  hipMemcpyAsync(gpu_input_values,
                  input_data.data(),
                  input_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice,
                  stream);
  T **gpu_output_values =
      reinterpret_cast<T **>(&gpu_input_values[input_data.size()]);
  hipMemcpyAsync(gpu_output_values,
                  output_data.data(),
                  output_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice,
                  stream);

  size_t N = static_cast<size_t>(batch_size * slot_num * embedding_size *
                                 embedx_concate_size);
  // first sum pool
  if (embedx_concate_size == 1) {
    if (need_filter) {  // filter
      FusedSeqpoolWithConvKernelFilter<<<GET_BLOCK(N),
                                         PADDLE_CUDA_NUM_THREADS,
                                         0,
                                         stream>>>(N,
                                                   gpu_input_values,
                                                   gpu_seqpool_output_values,
                                                   lods_values,
                                                   batch_size,
                                                   embedding_size,
                                                   padding_value,
                                                   show_coeff,
                                                   clk_coeff,
                                                   threshold);
    } else {  // normal
      FusedSeqpoolWithConvKernelNormal<<<GET_BLOCK(N),
                                         PADDLE_CUDA_NUM_THREADS,
                                         0,
                                         stream>>>(N,
                                                   gpu_input_values,
                                                   gpu_seqpool_output_values,
                                                   lods_values,
                                                   batch_size,
                                                   embedding_size,
                                                   padding_value);
    }
  } else {
    FusedSeqpoolWithConvKernelNormalEmbedxConcate<<<GET_BLOCK(N),
                                                    PADDLE_CUDA_NUM_THREADS,
                                                    0,
                                                    stream>>>(
        N,
        gpu_input_values,
        gpu_seqpool_output_values,
        lods_values,
        batch_size,
        embedding_size,
        padding_value,
        embedx_concate_size);
  }
  // second log
  if (use_cvm) {
    if (show_filter) {
      N = static_cast<size_t>(batch_size * slot_num * (embedding_size - 1) *
                              embedx_concate_size);
      if (embedx_concate_size == 1) {
        FusedCVMWithConvKernelWithOutShow<<<GET_BLOCK(N),
                                            PADDLE_CUDA_NUM_THREADS,
                                            0,
                                            stream>>>(N,
                                                      gpu_output_values,
                                                      gpu_seqpool_output_values,
                                                      batch_size,
                                                      embedding_size,
                                                      embedding_size - 1);
      } else {
        FusedCVMWithConvKernelWithOutShowConcate<<<GET_BLOCK(N),
                                                   PADDLE_CUDA_NUM_THREADS,
                                                   0,
                                                   stream>>>(
            N,
            gpu_output_values,
            gpu_seqpool_output_values,
            batch_size,
            embedding_size,
            embedding_size - 1,
            embedx_concate_size);
      }
    } else {
      if (embedx_concate_size == 1) {
        FusedCVMWithConvKernelNormal<<<GET_BLOCK(N),
                                       PADDLE_CUDA_NUM_THREADS,
                                       0,
                                       stream>>>(N,
                                                 gpu_output_values,
                                                 gpu_seqpool_output_values,
                                                 batch_size,
                                                 embedding_size);
      } else {
        FusedCVMWithConvKernelNormalConcate<<<GET_BLOCK(N),
                                              PADDLE_CUDA_NUM_THREADS,
                                              0,
                                              stream>>>(
            N,
            gpu_output_values,
            gpu_seqpool_output_values,
            batch_size,
            embedding_size,
            embedx_concate_size);
      }
    }
  } else {
    // not need show click input
    N = static_cast<size_t>(batch_size * slot_num *
                            (embedding_size - cvm_offset) *
                            embedx_concate_size);
    if (embedx_concate_size == 1) {
      FusedCVMWithConvKernelNoCVM<<<GET_BLOCK(N),
                                    PADDLE_CUDA_NUM_THREADS,
                                    0,
                                    stream>>>(N,
                                              gpu_output_values,
                                              gpu_seqpool_output_values,
                                              batch_size,
                                              (embedding_size - cvm_offset),
                                              cvm_offset);
    } else {
      FusedCVMWithConvKernelNoCVMConcate<<<GET_BLOCK(N),
                                           PADDLE_CUDA_NUM_THREADS,
                                           0,
                                           stream>>>(
          N,
          gpu_output_values,
          gpu_seqpool_output_values,
          batch_size,
          (embedding_size - cvm_offset),
          cvm_offset,
          embedx_concate_size);
    }
  }
}

// join grad
template <typename T>
__global__ void FusedSeqpoolCVMWithConvGradKernelWithCVM(
    const size_t N,
    T **out_grads_values,
    T **in_grads_values,
    T **cvm_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;  // embedx offset
    int x = key / batch_size;         // slot id
    int y = key % batch_size;         // ins id

    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] + y * embedding_size + offset);

    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];
    for (auto k = start; k < end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// join grad, expand slot's feasign
template <typename T>
__global__ void FusedSeqpoolCVMWithConvGradKernelWithCVMConcate(
    const size_t N,
    T **out_grads_values,
    T **in_grads_values,
    T **cvm_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const int cvm_offset,
    const int embedx_concate_size) {
  int concat_embedding_size = embedding_size * embedx_concate_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int concate_index = concat_offset / embedding_size;  // concat id
    int offset = concat_offset % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];
    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] + y * concat_embedding_size +
                     embedding_size * concate_index + offset);

    auto concat_end = start + concate_index + 1;
    if (concat_end > end || concate_index == embedx_concate_size - 1) {
      concat_end = end;
    }
    for (auto k = start + concate_index; k < concat_end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// join only show not has click
template <typename T>
__global__ void FusedSeqpoolCVMWithConvGradKernelWithShow(
    const size_t N,
    T **out_grads_values,
    T **in_grads_values,
    T **cvm_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;  // embedx offset
    int x = key / batch_size;         // slot id
    int y = key % batch_size;         // ins id

    T &val =
        (offset < cvm_offset)
            ? *(cvm_values[x] + y * cvm_offset + offset)
            : *(out_grads_values[x] + y * (embedding_size - 1) + offset - 1);

    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];
    for (auto k = start; k < end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// join only show not has click, expand slot's feasign
template <typename T>
__global__ void FusedSeqpoolCVMWithConvGradKernelWithShowConcate(
    const size_t N,
    T **out_grads_values,
    T **in_grads_values,
    T **cvm_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const int cvm_offset,
    const int embedx_concate_size) {
  int concat_embedding_size = embedding_size * embedx_concate_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int concate_index = concat_offset / embedding_size;  // concat id
    int offset = concat_offset % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];

    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] +
                     y * (embedding_size - 1) * embedx_concate_size +
                     (embedding_size - 1) * concate_index + offset - 1);
    auto concat_end = start + concate_index + 1;
    if (concat_end > end || concate_index == embedx_concate_size - 1) {
      concat_end = end;
    }
    for (auto k = start + concate_index; k < concat_end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// update grad
template <typename T>
__global__ void FusedSeqpoolCVMWithConvGradKernelNoCVM(
    const size_t N,
    T **out_grads_values,
    T **in_grads_values,
    T **cvm_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;  // embedx offset
    int x = key / batch_size;         // slot id
    int y = key % batch_size;         // ins id

    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] + y * (embedding_size - cvm_offset) +
                     offset - cvm_offset);

    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];
    for (auto k = start; k < end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// update grad, expand slot's feasign
template <typename T>
__global__ void FusedSeqpoolCVMWithConvGradKernelNoCVMConcate(
    const size_t N,
    T **out_grads_values,
    T **in_grads_values,
    T **cvm_values,
    const size_t *lods_values,
    const int batch_size,
    const int embedding_size,
    const int cvm_offset,
    const int embedx_concate_size) {
  int concat_embedding_size = embedding_size * embedx_concate_size;
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / concat_embedding_size;
    int concat_offset = i % concat_embedding_size;
    int concate_index = concat_offset / embedding_size;  // concat id
    int offset = concat_offset % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id

    auto &start = lods_values[x * (batch_size + 1) + y];
    auto &end = lods_values[x * (batch_size + 1) + y + 1];

    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] +
                     y * (embedding_size - cvm_offset) * embedx_concate_size +
                     (embedding_size - cvm_offset) * concate_index + offset -
                     cvm_offset);
    auto concat_end = start + concate_index + 1;
    if (concat_end > end || concate_index == embedx_concate_size - 1) {
      concat_end = end;
    }
    for (auto k = start + concate_index; k < concat_end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

template <typename T>
void FusedSeqpoolCVMGradWithConv(const paddle::platform::Place &place,
                                 const std::vector<const T *> &out_grads_data,
                                 const std::vector<T *> &in_grads_data,
                                 const std::vector<const T *> &cvm_data,
                                 const size_t *lods_values,
                                 const int batch_size,
                                 const int slot_num,
                                 const int embedding_size,
                                 const bool use_cvm,
                                 const int cvm_offset,
                                 bool show_filter,
                                 const int embedx_concate_size) {
  auto stream = dynamic_cast<phi::GPUContext *>(
                    platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  size_t total_ptr_len =
      out_grads_data.size() + in_grads_data.size() + cvm_data.size();
  auto temp_ptr = memory::AllocShared(place, total_ptr_len * sizeof(void *));
  T **gpu_out_grads_values = reinterpret_cast<T **>(temp_ptr->ptr());
  hipMemcpyAsync(gpu_out_grads_values,
                  out_grads_data.data(),
                  out_grads_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice,
                  stream);

  T **gpu_in_grads_values =
      reinterpret_cast<T **>(&gpu_out_grads_values[out_grads_data.size()]);
  hipMemcpyAsync(gpu_in_grads_values,
                  in_grads_data.data(),
                  in_grads_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice,
                  stream);

  T **gpu_cvm_values =
      reinterpret_cast<T **>(&gpu_in_grads_values[in_grads_data.size()]);
  hipMemcpyAsync(gpu_cvm_values,
                  cvm_data.data(),
                  cvm_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice,
                  stream);

  size_t N = static_cast<size_t>(batch_size * slot_num * embedding_size *
                                 embedx_concate_size);
  if (use_cvm) {
    if (show_filter) {
      if (embedx_concate_size == 1) {
        FusedSeqpoolCVMWithConvGradKernelWithShow<<<GET_BLOCK(N),
                                                    PADDLE_CUDA_NUM_THREADS,
                                                    0,
                                                    stream>>>(
            N,
            gpu_out_grads_values,
            gpu_in_grads_values,
            gpu_cvm_values,
            lods_values,
            batch_size,
            embedding_size,
            cvm_offset);
      } else {
        FusedSeqpoolCVMWithConvGradKernelWithShowConcate<<<
            GET_BLOCK(N),
            PADDLE_CUDA_NUM_THREADS,
            0,
            stream>>>(N,
                      gpu_out_grads_values,
                      gpu_in_grads_values,
                      gpu_cvm_values,
                      lods_values,
                      batch_size,
                      embedding_size,
                      cvm_offset,
                      embedx_concate_size);
      }
    } else {
      if (embedx_concate_size == 1) {
        FusedSeqpoolCVMWithConvGradKernelWithCVM<<<GET_BLOCK(N),
                                                   PADDLE_CUDA_NUM_THREADS,
                                                   0,
                                                   stream>>>(
            N,
            gpu_out_grads_values,
            gpu_in_grads_values,
            gpu_cvm_values,
            lods_values,
            batch_size,
            embedding_size,
            cvm_offset);
      } else {
        FusedSeqpoolCVMWithConvGradKernelWithCVMConcate<<<
            GET_BLOCK(N),
            PADDLE_CUDA_NUM_THREADS,
            0,
            stream>>>(N,
                      gpu_out_grads_values,
                      gpu_in_grads_values,
                      gpu_cvm_values,
                      lods_values,
                      batch_size,
                      embedding_size,
                      cvm_offset,
                      embedx_concate_size);
      }
    }
  } else {
    // update grad
    if (embedx_concate_size == 1) {
      FusedSeqpoolCVMWithConvGradKernelNoCVM<<<GET_BLOCK(N),
                                               PADDLE_CUDA_NUM_THREADS,
                                               0,
                                               stream>>>(N,
                                                         gpu_out_grads_values,
                                                         gpu_in_grads_values,
                                                         gpu_cvm_values,
                                                         lods_values,
                                                         batch_size,
                                                         embedding_size,
                                                         cvm_offset);
    } else {
      FusedSeqpoolCVMWithConvGradKernelNoCVMConcate<<<GET_BLOCK(N),
                                                      PADDLE_CUDA_NUM_THREADS,
                                                      0,
                                                      stream>>>(
          N,
          gpu_out_grads_values,
          gpu_in_grads_values,
          gpu_cvm_values,
          lods_values,
          batch_size,
          embedding_size,
          cvm_offset,
          embedx_concate_size);
    }
  }
}

template <typename T>
class FusedSeqpoolCVMWithConvCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto inputs = ctx.MultiInput<LoDTensor>("X");
    auto outputs = ctx.MultiOutput<framework::Tensor>("Out");

    const int slot_size = static_cast<int>(inputs.size());
    std::vector<const float *> input_data(slot_size);
    std::vector<T *> output_data(slot_size);

    phi::DenseTensor seqpool_tensor;

    auto padding_value = ctx.Attr<float>("pad_value");
    auto use_cvm = ctx.Attr<bool>("use_cvm");
    bool need_filter = ctx.Attr<bool>("need_filter");
    float show_coeff = ctx.Attr<float>("show_coeff");
    float clk_coeff = ctx.Attr<float>("clk_coeff");
    float threshold = ctx.Attr<float>("threshold");
    const int cvm_offset = ctx.Attr<int>("cvm_offset");
    bool show_filter = ctx.Attr<bool>("show_filter");
    const int embedx_concate_size = ctx.Attr<int>("embedx_concate_size");
    CHECK(embedx_concate_size > 0) << "embedx_concate_size must more than 1";

    auto place = ctx.GetPlace();

    CHECK(inputs[0]->dims()[0] > 0);
    int embedding_size = inputs[0]->numel() / inputs[0]->dims()[0];
    int batch_size = inputs[0]->lod()[0].size() - 1;

    T *seqpool_ptr = seqpool_tensor.mutable_data<T>(
        {slot_size * batch_size, embedding_size * embedx_concate_size}, place);
    // lod ptr
    auto lods_values = memory::AllocShared(
        place, sizeof(size_t) * slot_size * (batch_size + 1));
    size_t *lods_ptr = reinterpret_cast<size_t *>(lods_values->ptr());

    auto stream = dynamic_cast<phi::GPUContext *>(
                      platform::DeviceContextPool::Instance().Get(place))
                      ->stream();
    for (int i = 0; i < slot_size; ++i) {
      const auto *input = inputs[i];

      CHECK(input->lod().size() == 1);
      auto lod_data = input->lod()[0];
      int cur_batch = lod_data.size() - 1;
      CHECK(batch_size == cur_batch)
          << "batch: " << batch_size << ", current: " << cur_batch;

      input_data[i] = reinterpret_cast<const T *>(input->data<T>());
      auto *output = outputs[i];
      if (use_cvm) {
        if (show_filter) {
          // show will filtered
          output->Resize(
              {batch_size, (embedding_size - 1) * embedx_concate_size});
        } else {
          // show will filtered
          output->Resize({batch_size, embedding_size * embedx_concate_size});
        }
      } else {
        output->Resize(
            {batch_size, (embedding_size - cvm_offset) * embedx_concate_size});
      }
      output_data[i] = reinterpret_cast<T *>(output->mutable_data<T>(place));
      // copy load to gpu
      hipMemcpyAsync(&lods_ptr[(batch_size + 1) * i],
                      lod_data.data(),
                      lod_data.size() * sizeof(size_t),
                      hipMemcpyHostToDevice,
                      stream);
    }
    FusedSeqpoolCVMWithConv(place,
                            input_data,
                            output_data,
                            seqpool_ptr,
                            lods_ptr,
                            batch_size,
                            slot_size,
                            embedding_size,
                            padding_value,
                            use_cvm,
                            need_filter,
                            show_coeff,
                            clk_coeff,
                            threshold,
                            cvm_offset,
                            show_filter,
                            embedx_concate_size);
  }
};

template <typename T>
class FusedSeqpoolCVMWithConvGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto out_grads = ctx.MultiInput<LoDTensor>(framework::GradVarName("Out"));
    auto in_grads = ctx.MultiOutput<LoDTensor>(framework::GradVarName("X"));
    auto *cvm = ctx.Input<LoDTensor>("CVM");

    std::string pooltype = ctx.Attr<std::string>("pooltype");
    auto use_cvm = ctx.Attr<bool>("use_cvm");
    const int cvm_offset = ctx.Attr<int>("cvm_offset");
    bool show_filter = ctx.Attr<bool>("show_filter");
    const int embedx_concate_size = ctx.Attr<int>("embedx_concate_size");

    const int slot_size = static_cast<int>(in_grads.size());
    std::vector<const T *> out_grads_data(slot_size);
    std::vector<T *> in_grads_data(slot_size);
    std::vector<const T *> cvm_data(slot_size);

    auto place = ctx.GetPlace();

    CHECK(in_grads[0]->dims()[0] > 0);
    int embedding_size = in_grads[0]->numel() / in_grads[0]->dims()[0];
    int batch_size = in_grads[0]->lod()[0].size() - 1;

    // lod ptr
    auto lods_values = memory::AllocShared(
        place, sizeof(size_t) * slot_size * (batch_size + 1));
    size_t *lods_ptr = reinterpret_cast<size_t *>(lods_values->ptr());
    auto stream = dynamic_cast<phi::GPUContext *>(
                      platform::DeviceContextPool::Instance().Get(place))
                      ->stream();

    for (int i = 0; i < slot_size; ++i) {
      auto *in_grad = in_grads[i];

      auto lod_data = in_grad->lod()[0];
      int cur_batch = lod_data.size() - 1;
      CHECK(batch_size == cur_batch)
          << "batch: " << batch_size << ", current: " << cur_batch;
      auto *out_grad = out_grads[i];
      out_grads_data[i] = reinterpret_cast<const T *>(out_grad->data<T>());

      in_grads_data[i] = reinterpret_cast<T *>(in_grad->mutable_data<T>(place));
      // copy load to gpu
      hipMemcpyAsync(&lods_ptr[(batch_size + 1) * i],
                      lod_data.data(),
                      lod_data.size() * sizeof(size_t),
                      hipMemcpyHostToDevice,
                      stream);
      cvm_data[i] = reinterpret_cast<const T *>(cvm->data<T>());
    }
    FusedSeqpoolCVMGradWithConv(place,
                                out_grads_data,
                                in_grads_data,
                                cvm_data,
                                lods_ptr,
                                batch_size,
                                slot_size,
                                embedding_size,
                                use_cvm,
                                cvm_offset,
                                show_filter,
                                embedx_concate_size);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(fused_seqpool_cvm_with_conv,
                        ops::FusedSeqpoolCVMWithConvCUDAKernel<float>);

REGISTER_OP_CUDA_KERNEL(fused_seqpool_cvm_with_conv_grad,
                        ops::FusedSeqpoolCVMWithConvGradCUDAKernel<float>);
